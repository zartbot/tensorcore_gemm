#include "common.hpp"
#include "hipblas.h"

void launch_gemm(size_t M, size_t N, size_t K, half *A, half *B, half *C, half alpha, half beta)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, K, A,
                 HIP_R_16F, K, &beta, C, HIP_R_16F, N, HIPBLAS_COMPUTE_16F,
                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


int main()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    testError(launch_gemm,0);
    perf_measure(launch_gemm);

}